
#include <hip/hip_runtime.h>
extern "C"
__global__ void capByScalar(int n, float *a, float b, float *result)
{
	float cap = b;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] < cap ? a[i] : cap;
    }
}

extern "C"
__global__ void floorByScalar(int n, float *a, float b, float *result)
{
	float floor = b;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] > floor ? a[i] : floor;
    }
}

extern "C"
__global__ void addScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] + b;
    }
}

extern "C"
__global__ void subScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] - b;
    }
}

extern "C"
__global__ void busScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = -a[i] + b;
    }
}

extern "C"
__global__ void multScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] * b;
    }

}

extern "C"
__global__ void divScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] / b;
    }

}

extern "C"
__global__ void vidScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = b / a[i];
    }

}

extern "C"
__global__ void squared(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] * a[i];
    }

}

extern "C"
__global__ void cuPow(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = pow(a[i],b);
    }

}

extern "C"
__global__ void cuSqrt(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = sqrt(a[i]);
    }
}

extern "C"
__global__ void cuExp(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = (float)exp((double)a[i]);
    }
}

extern "C"
__global__ void cuLog(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = (float)log((double)a[i]);
    }
}

extern "C"
__global__ void invert(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = 1.0f / a[i];
    }
}

extern "C"
__global__ void cuAbs(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = abs(a[i]);
    }
}


extern "C"
__global__ void cap(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] < b[i] ? a[i] : b[i];
    }
}

extern "C"
__global__ void cuFloor(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] > b[i] ? a[i] : b[i];
    }
}

extern "C"
__global__ void add(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] + b[i];
    }

}

extern "C"
__global__ void sub(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] - b[i];
    }

}

extern "C"
__global__ void mult(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] * b[i];
    }

}

extern "C"
__global__ void cuDiv(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] / b[i];
    }

}

extern "C"
__global__ void accrue(int n, float *a, float *b, float p, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] * (1.0f + b[i] * p);
    }
}

extern "C"
__global__ void discount(int n, float *a, float *b, float p, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        // We force to avoid fma
        float prod = b[i] * p;
        float fma = (1.0f + prod);
        result[i] = a[i] / fma;
    }
}

extern "C"
__global__ void addProduct(int n, float *a, float *b, float *c, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] + b[i] * c[i];
    }
}

extern "C"
__global__ void addProduct_vs(int n, float *a, float *b, float c, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] + b[i] * c;
    }
}

extern "C"
__global__ void addRatio(int n, float *a, float *b, float *c, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] + b[i] / c[i];
    }
}

extern "C"
__global__ void subRatio(int n, float *a, float *b, float *c, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] - b[i] / c[i];
    }
}

extern "C"
__global__ void reduceFloatVectorToDoubleScalar(int size, void *data, double *result) {
	float *fdata = (float*) data;

	extern __shared__ double sdata[];
	double* s2data = sdata + blockDim.x;
	double* cdata = s2data + blockDim.x;

	// perform first level of reduction,
	// reading from global memory, writing to shared memory unsigned int tid = threadIdx.x;
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	sdata[tid] = (double)(i < size ? fdata[i] : 0) + (double)(i+blockDim.x < size ? fdata[i+blockDim.x] : 0);
	cdata[tid] = sdata[tid] - (double)(i < size ? fdata[i] : 0) - (double)(i+blockDim.x < size ? fdata[i+blockDim.x] : 0);
	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			s2data[tid] = sdata[tid] + sdata[tid + s] - cdata[tid] - cdata[tid+s];
			cdata[tid] = (s2data[tid] - sdata[tid]) - sdata[tid + s];
			sdata[tid] = s2data[tid];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) result[blockIdx.x] = sdata[0];
}

/*
 * Perfom a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
 */ 
extern "C"
__global__ void reducePartial(int size, void *data, void *result) {
	float *fdata = (float*) data;
	float *sum = (float*) result;

	extern __shared__ double sdata[];
	double* s2data = sdata + blockDim.x;
	double* cdata = s2data + blockDim.x;

	// perform first level of reduction,
	// reading from global memory, writing to shared memory unsigned int tid = threadIdx.x;
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	sdata[tid] = (double)(i < size ? fdata[i] : 0) + (double)(i+blockDim.x < size ? fdata[i+blockDim.x] : 0);
	cdata[tid] = sdata[tid] - (double)(i < size ? fdata[i] : 0) - (double)(i+blockDim.x < size ? fdata[i+blockDim.x] : 0);
	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			s2data[tid] = sdata[tid] + sdata[tid + s] - cdata[tid] - cdata[tid+s];
			cdata[tid] = (s2data[tid] - sdata[tid]) - sdata[tid + s];
			sdata[tid] = s2data[tid];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) sum[blockIdx.x] = sdata[0];
}
